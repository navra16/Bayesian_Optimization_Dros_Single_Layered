#include "hip/hip_runtime.h"
#include "StrainTensor.h"
#include <algorithm>   // for std::max
#include <cmath>
#include "System.h"

// This function updates each edge's rest length based on a strain tensor
// by decomposing the edge vector into radial and tangential components.
// The radial component is scaled by (1 + epsilon_r) and the tangential component
// by (1 + epsilon_t), and then they are recombined to form the new rest length.
void applyStrainToEdges( 
    GeneralParams& generalParams,
    CoordInfoVecs& coordInfoVecs,
    LinearSpringInfoVecs& linearSpringInfoVecs)

//double epsilon_r, double epsilon_t,
//                        const std::vector<double>& nodeLocX,
//                        const std::vector<double>& nodeLocY,
//                        const std::vector<int>& edges2Nodes_1,
//                        const std::vector<int>& edges2Nodes_2,
//                        double centerX, double centerY,
//                        std::vector<double>& edgeRestLengths)
{
    int numEdges = coordInfoVecs.edges2Nodes_1.size();
    for (int e = 0; e < numEdges; ++e) {
        int i = coordInfoVecs.edges2Nodes_1[e];
        int j = coordInfoVecs.edges2Nodes_2[e];
        double x1 = coordInfoVecs.nodeLocX[i], y1 = coordInfoVecs.nodeLocY[i];
        double x2 = coordInfoVecs.nodeLocX[j], y2 = coordInfoVecs.nodeLocY[j];

        // Compute the midpoint of the edge.
        double midX = 0.5 * (x1 + x2);
        double midY = 0.5 * (y1 + y2);

        // Compute the radial vector from the center to the midpoint.
        double rx = midX - generalParams.centerX;
        double ry = midY - generalParams.centerY;
        double r_norm = sqrt(rx * rx + ry * ry);
        double urx = (r_norm > 1e-12) ? rx / r_norm : 0.0;
        double ury = (r_norm > 1e-12) ? ry / r_norm : 0.0;

        // Compute the edge vector.
        double dx = x2 - x1;
        double dy = y2 - y1;
        double L = sqrt(dx * dx + dy * dy);

        // Decompose the edge vector into radial and tangential components.
        double L_r = dx * urx + dy * ury;
        // Use std::max to avoid negative values under the square root.
        double L_t = sqrt(std::max(0.0, L * L - L_r * L_r));

        // Apply the strain: scale radial and tangential components.
        double new_L_r = L_r * (1.0 + generalParams.epsilon_r);
        double new_L_t = L_t * (1.0 + generalParams.epsilon_t);

        // Recombine the components to compute the new rest length.
        double newRestLength = sqrt(new_L_r * new_L_r + new_L_t * new_L_t);

        // Update the edge's rest length in the host vector.
        linearSpringInfoVecs.edge_initial_length[e] = newRestLength;
    }
}
