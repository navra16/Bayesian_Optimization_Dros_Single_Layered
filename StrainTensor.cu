#include "hip/hip_runtime.h"
#include "StrainTensor.h"
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <cmath>
#include <algorithm>

void applyStrainToEdges(GeneralParams& generalParams,
                        CoordInfoVecs& coordInfoVecs,
                        LinearSpringInfoVecs& linearSpringInfoVecs)
{
    int numEdges = coordInfoVecs.edges2Nodes_1.size();
    // Set up the functor with the strain parameters and disc geometry.
    StrainTensorFunctor functor(
        generalParams.epsilon_r_center,   // radial strain at center
        generalParams.epsilon_r_edge,     // radial strain at boundary
        generalParams.epsilon_t,          // tangential strain (uniform)
        generalParams.centerX,            // disc center X
        generalParams.centerY,            // disc center Y
        generalParams.disc_radius,        // disc radius
        thrust::raw_pointer_cast(coordInfoVecs.nodeLocX.data()),
        thrust::raw_pointer_cast(coordInfoVecs.nodeLocY.data()),
        thrust::raw_pointer_cast(coordInfoVecs.edges2Nodes_1.data()),
        thrust::raw_pointer_cast(coordInfoVecs.edges2Nodes_2.data()),
        // We assume that linearSpringInfoVecs.edge_rest_length has been allocated and
        // initially set (typically to the initial lengths of the edges)
        thrust::raw_pointer_cast(linearSpringInfoVecs.edge_rest_length.data())
    );

    thrust::for_each(thrust::device,
                     thrust::counting_iterator<int>(0),
                     thrust::counting_iterator<int>(numEdges),
                     functor);
}


//#include "StrainTensor.h"
//#include <thrust/for_each.h>
//#include <thrust/iterator/counting_iterator.h>
//#include <cmath>
//#include <algorithm>
//
//void applyStrainToEdges(GeneralParams& generalParams,
//                        CoordInfoVecs& coordInfoVecs,
//                        LinearSpringInfoVecs& linearSpringInfoVecs)
//{
//    int numEdges = coordInfoVecs.edges2Nodes_1.size();
//    // Set up the functor with current strain values and center.
//    StrainTensorFunctor functor(generalParams.epsilon_r,
//                                 generalParams.epsilon_t,
//                                 generalParams.centerX,
//                                 generalParams.centerY,
//                                 thrust::raw_pointer_cast(coordInfoVecs.nodeLocX.data()),
//                                 thrust::raw_pointer_cast(coordInfoVecs.nodeLocY.data()),
//                                 thrust::raw_pointer_cast(coordInfoVecs.edges2Nodes_1.data()),
//                                 thrust::raw_pointer_cast(coordInfoVecs.edges2Nodes_2.data()),
//                                 thrust::raw_pointer_cast(linearSpringInfoVecs.edge_initial_length.data()));
//    thrust::for_each(thrust::device,
//                     thrust::counting_iterator<int>(0),
//                     thrust::counting_iterator<int>(numEdges),
//                     functor);
//}
//
//
////#include "StrainTensor.h"
////#include <algorithm>   
////#include <cmath>
////#include "System.h"
////
//////This function updates each edge's rest length based on a strain tensor
//////by decomposing the edge vector into radial and tangential components.
//////The radial component is scaled by (1 + epsilon_r) and the tangential component
//////by (1 + epsilon_t), and then they are recombined to form the new rest length.
////void applyStrainToEdges( 
////    GeneralParams& generalParams,
////    CoordInfoVecs& coordInfoVecs,
////    LinearSpringInfoVecs& linearSpringInfoVecs)
////
//////double epsilon_r, double epsilon_t,
//////                        const std::vector<double>& nodeLocX,
//////                        const std::vector<double>& nodeLocY,
//////                        const std::vector<int>& edges2Nodes_1,
//////                        const std::vector<int>& edges2Nodes_2,
//////                        double centerX, double centerY,
//////                        std::vector<double>& edgeRestLengths)
////{
////    int numEdges = coordInfoVecs.edges2Nodes_1.size();
////    for (int e = 0; e < numEdges; ++e) {
////        int i = coordInfoVecs.edges2Nodes_1[e];
////        int j = coordInfoVecs.edges2Nodes_2[e];
////        double x1 = coordInfoVecs.nodeLocX[i], y1 = coordInfoVecs.nodeLocY[i];
////        double x2 = coordInfoVecs.nodeLocX[j], y2 = coordInfoVecs.nodeLocY[j];
////
////        // Compute the midpoint of the edge.
////        double midX = 0.5 * (x1 + x2);
////        double midY = 0.5 * (y1 + y2);
////
////        // Compute the radial vector from the center to the midpoint.
////        double rx = midX - generalParams.centerX;
////        double ry = midY - generalParams.centerY;
////        double r_norm = sqrt(rx * rx + ry * ry);
////        double urx = (r_norm > 1e-12) ? rx / r_norm : 0.0;
////        double ury = (r_norm > 1e-12) ? ry / r_norm : 0.0;
////
////        // Compute the edge vector.
////        double dx = x2 - x1;
////        double dy = y2 - y1;
////        double L = sqrt(dx * dx + dy * dy);
////
////        // Decompose the edge vector into radial and tangential components.
////        double L_r = dx * urx + dy * ury;
////        // Use std::max to avoid negative values under the square root.
////        double L_t = sqrt(std::max(0.0, L * L - L_r * L_r));
////
////        // Apply the strain: scale radial and tangential components.
////        double new_L_r = L_r * (1.0 + generalParams.epsilon_r);
////        double new_L_t = L_t * (1.0 + generalParams.epsilon_t);
////
////        // Recombine the components to compute the new rest length.
////        double newRestLength = sqrt(new_L_r * new_L_r + new_L_t * new_L_t);
////
////        // Update the edge's rest length in the host vector.
////        linearSpringInfoVecs.edge_rest_length[e] = newRestLength;
////    }
////}
////// The problem with the above tensor is, that while the edges of the springs are being computed and updated, the directionality of that new length is yet to be determined. So in combination with the other updated edges, the whole system is being thrown out of whack. What I need to do is have a strain tensor that is updating the lenths by veeeeryy little. Currently it is 0.05, so 5%. Seems to be a lot idk. 