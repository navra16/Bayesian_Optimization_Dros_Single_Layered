#include "hip/hip_runtime.h"
#include "SegmentVolumeConstraint.h"
#include <cmath>
#include <iostream>

// Helper function: Compute the volume of a closed surface using the divergence theorem,
// summing contributions from each triangle in the segment.
// Here we assume each triangle contributes: V_triangle = (1/6) * (x1*(y2*z3 - y3*z2) + ...).
// For a non-closed surface, this is an approximation.
double computeSegmentVolume(const CoordInfoVecs& coordInfoVecs,
                            const std::vector<int>& segmentTriangleIndices) {
    double volume = 0.0;
    int numTriangles = segmentTriangleIndices.size();
    for (int idx = 0; idx < numTriangles; ++idx) {
        int t = segmentTriangleIndices[idx];
        int i = coordInfoVecs.triangles2Nodes_1[t];
        int j = coordInfoVecs.triangles2Nodes_2[t];
        int k = coordInfoVecs.triangles2Nodes_3[t];
        double x1 = coordInfoVecs.nodeLocX[i], y1 = coordInfoVecs.nodeLocY[i], z1 = coordInfoVecs.nodeLocZ[i];
        double x2 = coordInfoVecs.nodeLocX[j], y2 = coordInfoVecs.nodeLocY[j], z2 = coordInfoVecs.nodeLocZ[j];
        double x3 = coordInfoVecs.nodeLocX[k], y3 = coordInfoVecs.nodeLocY[k], z3 = coordInfoVecs.nodeLocZ[k];
        double vol = (x1 * (y2 * z3 - y3 * z2) +
                      x2 * (y3 * z1 - y1 * z3) +
                      x3 * (y1 * z2 - y2 * z1)) / 6.0;
        volume += vol;
    }
    return fabs(volume);
}

void ApplySegmentVolumeConstraint(GeneralParams& generalParams,
                                  CoordInfoVecs& coordInfoVecs,
                                  const std::vector<int>& segmentTriangleIndices,
                                  const std::vector<int>& segmentNodeIndices)
{
    // Compute the current volume of the segment.
    double currentVolume = computeSegmentVolume(coordInfoVecs, segmentTriangleIndices);
    // Use the equilibrium volume from generalParams (or define a new parameter if desired).
    double eqVolume = generalParams.eq_total_volume;  
    // Compute volume error.
    double volumeError = currentVolume - eqVolume;
    
    // Use a volume spring constant (assumed stored in generalParams.volume_spring_constant) to determine the corrective force.
    double K_volume_segment = generalParams.volume_spring_constant;
    double correctiveForce = K_volume_segment * volumeError;
    
    // For simplicity, apply the corrective force uniformly to all nodes in the segment in the Z direction.
    int numNodes = segmentNodeIndices.size();
    if (numNodes == 0) return;
    double forcePerNode = correctiveForce / numNodes;
    
    // Update the Z-component of the node force for each node in the segment.
    for (int n : segmentNodeIndices) {
        // Note: this assumes that coordInfoVecs.nodeForceZ is accessible on host.
        coordInfoVecs.nodeForceZ[n] -= forcePerNode;
    }
    
    std::cout << "Segment volume: " << currentVolume 
              << " (eq: " << eqVolume << "), applied corrective force (Z): " << correctiveForce << std::endl;
}
